#include "hip/hip_runtime.h"

#include "test.h"

__global__ void add(float a, float b, float *c)
{
  *c = a + b;
}

void test_add(float a, float b, float &c)
{
  float *dev_c;
  hipMalloc(&dev_c, sizeof(float));

  add<<<1, 1>>>(a, b, dev_c);

  // Copy vectors to device
  hipMemcpy(&c, dev_c, sizeof(float), hipMemcpyDeviceToHost);

  // Copy results back from device.
  std::printf("%f + %f = %f \n", a, b, c);

  hipFree(dev_c);
}
