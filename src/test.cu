#include "hip/hip_runtime.h"

#include "test.h"

__global__ void add(float a, float b, float *c)
{
  *c = a + b;
}

__global__ void array_add(float *a, float *b, float *c, int N)
{
  int tid = blockIdx.x;
  if (tid < N)
  {
    c[tid] = a[tid] + b[tid];
  }
}

void test_add(float a, float b)
{
  // prefix dev_ : device
  float c;
  float *dev_c;
  hipMalloc(&dev_c, sizeof(float));

  add<<<1, 1>>>(a, b, dev_c);

  hipMemcpy(&c, dev_c, sizeof(float), hipMemcpyDeviceToHost);

  // Copy results back from device.
  std::printf("%f + %f = %f \n", a, b, c);

  hipFree(dev_c);
}

void test_array_add(float *a, float *b, int N)
{

  float c[N];
  float *dev_a, *dev_b, *dev_c;
  hipMalloc(&dev_a, sizeof(float) * N);
  hipMalloc(&dev_b, sizeof(float) * N);
  hipMalloc(&dev_c, sizeof(float) * N);

  hipMemcpy(dev_a, a, sizeof(a), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof(b), hipMemcpyHostToDevice);

  array_add<<<N, 1>>>(dev_a, dev_b, dev_c, N);

  hipMemcpy(c, dev_c, sizeof(c), hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  for (int i = 0; i < N; i++)
  {
    std::printf("%f + %f = %f \n", a[i], b[i], c[i]);
  }
}